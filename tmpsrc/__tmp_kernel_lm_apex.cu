#include "hip/hip_runtime.h"
#include "./layer_norm/ln.h"
#include "./layer_norm/ln_utils.cuh"
#include "./layer_norm/ln_kernel_traits.h"
using namespace layer_norm;
template <typename T, int N>
struct Tensor {
  __device__ T& operator[](int ind) {
    return data[ind];
  };

  T* data;
  int size[N];
  int stride[N];
};

__global__ void kernel1(Tensor<__half, 2> T0, Tensor<__half, 1> T1, Tensor<__half, 1> T2, Tensor<__half, 2> T20, Tensor<float, 2> T29, Tensor<float, 2> T31)
{
    FwdParams params;
    params.mu = &T29[0];
    params.rs = &T31[0];
    params.gamma = &T1[0];
    params.beta = &T2[0];
    params.x = &T0[0];

    using Ktraits = Kernel_traits<__half,
                                        __half,
                                        __half,
                                        float,
                                        int,
                                        10240,
                                        1,
                                        1,
                                        4,
                                        16
                                        >;
    enum { ROWS_PER_CTA = Ktraits::ROWS_PER_CTA };
    enum { WARPS_N = Ktraits::WARPS_N };
    enum { WARPS_M = Ktraits::WARPS_M };
    enum { THREADS_PER_ROW = Ktraits::THREADS_PER_ROW };
    enum { VEC_COLS_PER_LDG = Ktraits::VEC_COLS_PER_LDG };
    enum { BYTES_PER_ROW = Ktraits::BYTES_PER_ROW };
    enum { LDGS = Ktraits::LDGS };
    enum { NUM_ELTS = Ktraits::NUM_ELTS };
    enum { CTAS_PER_ROW = Ktraits::CTAS_PER_ROW };

    using output_t = typename Ktraits::output_t;
    using index_t = typename Ktraits::index_t;
    using compute_t = typename Ktraits::compute_t;
    using Ivec = typename Ktraits::Ivec;
    using Ovec = typename Ktraits::Ovec;
    using Wvec = typename Ktraits::Wvec;
    using Cvec = typename Ktraits::Cvec;

    using Stats = typename Ktraits::Stats;
    using stats_t = typename Stats::stats_t;

    extern __shared__ char smem_[];

    const index_t tidx = threadIdx.x;
    const index_t bidn = blockIdx.x % CTAS_PER_ROW;
    const index_t bidm = blockIdx.x / CTAS_PER_ROW;
    const index_t lane = tidx % THREADS_PER_WARP;
    const index_t warp = tidx / THREADS_PER_WARP;
    const index_t warp_m = warp / WARPS_N;
    const index_t warp_n = warp % WARPS_N;

    const index_t r = bidm * ROWS_PER_CTA + warp_m;
    const index_t c = bidn * THREADS_PER_ROW + warp_n * THREADS_PER_WARP + lane;

    Stats stats(params, bidm, bidn, warp_m, warp_n, lane, smem_);

    compute_t *mu_ptr = static_cast<compute_t *>(params.mu);
    compute_t *rs_ptr = static_cast<compute_t *>(params.rs);

    Wvec gamma[LDGS];
    Wvec beta[LDGS];
    index_t idx = c;
    #pragma unroll
    for( int it = 0; it < LDGS; it++ ) {
        gamma[it].load_from(params.gamma, idx);
        beta[it].load_from(params.beta, idx);
        idx += VEC_COLS_PER_LDG;
    }

    constexpr compute_t rn = 1.f / compute_t(Ktraits::COLS);

    for( int row = r; row < params.rows; row += params.ctas_per_col * ROWS_PER_CTA ) {
        Ivec x[LDGS];
        index_t idx = row * Ktraits::VEC_COLS + c;
        compute_t xf[LDGS * NUM_ELTS];
        #pragma unroll
        for( int it = 0; it < LDGS; it++ ) {
            x[it].load_from(params.x, idx);
            #pragma unroll
            for( int jt = 0; jt < NUM_ELTS; jt++ ) {
                compute_t x_ij = compute_t(x[it].data.elt[jt]);
                xf[it * NUM_ELTS + jt] =  x_ij;
            }
            idx += VEC_COLS_PER_LDG;
        }

        stats_t s = stats.compute(xf, rn);

        compute_t mu = layer_norm::Get<0>::of<stats_t, compute_t>(s);
        compute_t m2 = layer_norm::Get<1>::of<stats_t, compute_t>(s);

        if( bidn == 0 && warp_n == 0 && lane == 0 ) {
            mu_ptr[row] = mu;
        }

        compute_t rs = rsqrtf(rn * m2 + params.epsilon);

        if( bidn == 0 && warp_n == 0 && lane == 0 ) {
            rs_ptr[row] = rs;
        }

        Ovec z[LDGS];
        idx = row * Ktraits::VEC_COLS + c;
        #pragma unroll
        for( int it = 0; it < LDGS; it++ ) {
            #pragma unroll
            for( int jt = 0; jt < NUM_ELTS; jt++ ) {
                output_t y_ij = output_t(rs * (xf[it * NUM_ELTS + jt] - mu));
                output_t g_ij = gamma[it].data.elt[jt];
                output_t b_ij = beta[it].data.elt[jt];
                z[it].data.elt[jt] = (g_ij * y_ij + b_ij);
            }
            z[it].store_to(params.z, idx);
            idx += VEC_COLS_PER_LDG;
        }

    }
}
// Create forward launch function and register. Macro signature:
// HIDDEN_SIZE, WTYPE, ITYPE, OTYPE, CTYPE, CTAS_PER_ROW, WARPS_M, WARPS_N, BYTES_PER_LDG
// REGISTER_FWD_LAUNCHER(10240, fp16, fp16, fp16, fp32, 1, 1, 4, 16);
